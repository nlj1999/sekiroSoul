#include "hip/hip_runtime.h"

#include "sekiroSoul.h"

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(rtObject, top_object, , );


rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );
rtDeclareVariable(float3, bad_color, , );
rtBuffer<uchar4, 2>              output_buffer;

RT_PROGRAM void pinhole_camera()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	optix::Ray ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(top_object, ray, prd);

	output_buffer[launch_index] = make_color(prd.result);
}


rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void envmap_miss()
{
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v = 0.5f * (1.0f + sin(phi));
	prd_radiance.result = make_float3(tex2D(envmap, u, v));
}


RT_PROGRAM void any_hit_shadow()
{
	prd_shadow.attenuation = make_float3(0);

	rtTerminateRay();
}



rtDeclareVariable(float, metalKa, , ) = 1;
rtDeclareVariable(float, metalKs, , ) = 1;
rtDeclareVariable(float, metalroughness, , ) = .1;
rtDeclareVariable(float, rustKa, , ) = 1;
rtDeclareVariable(float, rustKd, , ) = 1;
rtDeclareVariable(float3, metalcolor, , ) = { .7, .7, .7 };
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<BasicLight>       lights;
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(float3, reflectivity_n, , );
#define MAXOCTAVES 6


RT_PROGRAM void metal_closest_hit_radiance()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 hit_point = ray.origin + t_hit * ray.direction;


	float3 color = metalcolor * metalKa * ambient_light_color;
	for (int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nmDl = dot(ffnormal, L);

		if (nmDl > 0.0f) {
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist);
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				float3 Lc = light.color * light_attenuation;
				color += rustKd * nmDl * Lc;

				float r = nmDl;
				if (nmDl > 0.0f) {
					float3 H = normalize(L - ray.direction);
					float nmDh = dot(ffnormal, H);
					if (nmDh > 0)
						color += r * metalKs * Lc * pow(nmDh, 1.f / metalroughness);
				}
			}
		}
	}

	float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n);
	float importance = prd_radiance.importance * optix::luminance(r);

	// reflection ray
	if (importance > importance_cutoff && prd_radiance.depth < max_depth) {
		PerRayData_radiance refl_prd;
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth + 1;
		float3 R = reflect(ray.direction, ffnormal);
		optix::Ray refl_ray(hit_point, R, RADIANCE_RAY_TYPE, scene_epsilon);
		rtTrace(top_object, refl_ray, refl_prd);
		color += r * refl_prd.result;
	}

	prd_radiance.result = color;
}


rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, reflectivity, , );
rtDeclareVariable(float3, tile_v0, , );
rtDeclareVariable(float3, tile_v1, , );
rtDeclareVariable(float3, crack_color, , );
rtDeclareVariable(float, crack_width, , );

RT_PROGRAM void floor_closest_hit_radiance()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	float v0 = dot(tile_v0, hit_point);
	float v1 = dot(tile_v1, hit_point);
	v0 = v0 - floor(v0);
	v1 = v1 - floor(v1);

	float3 local_Kd;
	if (v0 > crack_width && v1 > crack_width) {
		local_Kd = Kd;
	}
	else {
		local_Kd = crack_color;
	}

	for (int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if (nDl > 0.0f) {
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			optix::Ray shadow_ray(hit_point, L, SHADOW_RAY_TYPE, scene_epsilon, Ldist);
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if (fmaxf(light_attenuation) > 0.0f) {
				float3 Lc = light.color * light_attenuation;
				color += local_Kd * nDl * Lc;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
					color += Ks * Lc * pow(nDh, phong_exp);
			}

		}
	}

	float3 r = schlick(-dot(ffnormal, ray.direction), reflectivity_n);
	float importance = prd_radiance.importance * optix::luminance(r);

	// reflection ray
	if (importance > importance_cutoff && prd_radiance.depth < max_depth) {
		PerRayData_radiance refl_prd;
		refl_prd.importance = importance;
		refl_prd.depth = prd_radiance.depth + 1;
		float3 R = reflect(ray.direction, ffnormal);
		optix::Ray refl_ray(hit_point, R, RADIANCE_RAY_TYPE, scene_epsilon);
		rtTrace(top_object, refl_ray, refl_prd);
		color += r * refl_prd.result;
	}

	prd_radiance.result = color;
}

rtDeclareVariable(float3, particle_color, attribute particle_color, );
rtDeclareVariable(float, particle_opacity, attribute particle_opacity, );
rtDeclareVariable(float3, shadow_attenuation, , );


//
// Dielectric surface shader
//
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(float, fresnel_exponent, , );
rtDeclareVariable(float, fresnel_minimum, , );
rtDeclareVariable(float, fresnel_maximum, , );
rtDeclareVariable(float, refraction_index, , );
rtDeclareVariable(int, refraction_maxdepth, , );
rtDeclareVariable(int, reflection_maxdepth, , );
rtDeclareVariable(float3, refraction_color, , );
rtDeclareVariable(float3, reflection_color, , );
rtDeclareVariable(float3, extinction_constant, , );
RT_PROGRAM void fire_closest_hit_radiance()
{
	/*
	// intersection vectors
	const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
	const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
	const float3 i = ray.direction;                                            // incident direction

	float reflection = 1.0f;
	float3 result = make_float3(0.0);

	float3 beer_attenuation;
	if (dot(n, ray.direction) > 0) {
		// Beer's law attenuation
		beer_attenuation = exp(extinction_constant * t_hit);
	}
	else {
		beer_attenuation = make_float3(1);
	}

	// refraction
	if (prd_radiance.depth < min(refraction_maxdepth, max_depth))
	{
		float3 t;                                                            // transmission direction
		if (refract(t, i, n, refraction_index))
		{

			// check for external or internal reflection
			float cos_theta = dot(i, n);
			if (cos_theta < 0.0f)
				cos_theta = -cos_theta;
			else
				cos_theta = dot(t, n);

			reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

			float importance = prd_radiance.importance * (1.0f - reflection) * optix::luminance(refraction_color * beer_attenuation);
			if (importance > importance_cutoff) {
				optix::Ray ray(h, t, RADIANCE_RAY_TYPE, scene_epsilon);
				PerRayData_radiance refr_prd;
				refr_prd.depth = prd_radiance.depth + 1;
				refr_prd.importance = importance;

				rtTrace(top_object, ray, refr_prd);
				result += (1.0f - reflection) * refraction_color * refr_prd.result;
			}
			else {
				result += (1.0f - reflection) * refraction_color * cutoff_color;
			}
		}
		// else TIR
	}
	// reflection
	if (prd_radiance.depth < min(reflection_maxdepth, max_depth))
	{
		float3 r = reflect(i, n);

		float importance = prd_radiance.importance * reflection * optix::luminance(reflection_color * beer_attenuation);
		if (importance > importance_cutoff) {
			optix::Ray ray(h, r, RADIANCE_RAY_TYPE, scene_epsilon);
			PerRayData_radiance refl_prd;
			refl_prd.depth = prd_radiance.depth + 10;
			refl_prd.importance = importance;

			rtTrace(top_object, ray, refl_prd);
			result += reflection * reflection_color * refl_prd.result;
		}
		else {
			result += reflection * reflection_color * cutoff_color;
		}
	}

	result = result * beer_attenuation;
	prd_radiance.result = result;*/
	const float3 h = ray.origin + t_hit * ray.direction;            // hitpoint
	float opa = max(0.0f, particle_opacity);
	if (prd_radiance.depth < max_depth) {
		optix::Ray ray(h, ray.direction, RADIANCE_RAY_TYPE, scene_epsilon);
		PerRayData_radiance through;
		through.depth = prd_radiance.depth + 10;
		rtTrace(top_object, ray, through);
		prd_radiance.result = through.result * (1 - opa) + particle_color * opa;
	}
}

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}
